#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
//rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// panoramic camera
RT_PROGRAM void env_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf , M_PIf) + make_float2(M_PIf, 0);
  float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
  float3 ray_origin = eye;
  float3 ray_direction = normalize(angle.x*normalize(U) + angle.y*normalize(V) + angle.z*normalize(W));

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}