#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(optix::uint2, launch_index, rtLaunchIndex, );
rtBuffer<optix::float3, 2> output_buffer;

rtDeclareVariable(optix::float3, color_to_draw, , );

RT_PROGRAM void draw_solid_color()
{
	output_buffer[launch_index] = color_to_draw;
}