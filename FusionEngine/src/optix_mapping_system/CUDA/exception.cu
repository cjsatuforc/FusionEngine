#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}