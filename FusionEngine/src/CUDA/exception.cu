#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_uchar4( bad_color.x * 255, bad_color.y * 255, bad_color.z * 255, 255);
}